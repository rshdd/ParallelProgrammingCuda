#include "hip/hip_runtime.h"

#include "stdio.h"
#include "stdlib.h"
#include <iostream>

int main()
{
    int size = 1 << 25;
    int bytes = size * sizeof(float);

    // Allocate host memory
    float* h_a = (float*)malloc(bytes);

    // float* h_a;
    // cudaMallocHost((float**)&h_a, bytes);

    // Allocate device memory
    float* d_a;
    hipMalloc((float**)&d_a, bytes);

    // Initialize host memory
    for (int i = 0; i < size; i++) {
        h_a[i] = rand() % 10;
    }

    hipMemcpy(d_a, h_a, bytes, hipMemcpyHostToDevice);
    hipMemcpy(h_a, d_a, bytes, hipMemcpyDeviceToHost);

    // Free memory
    hipFree(d_a);
    free(h_a);
    // cudaFreeHost(h_a);
    hipDeviceReset();

    return EXIT_SUCCESS;
}