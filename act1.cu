#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>


__global__ void print_hello_cuda() {
    int i = threadIdx.x;
    printf("[DEVICE] ThreadIdx.x: %d\n", i);
}

int main()
{
    print_hello_cuda << < 2, 8 >> > ();
    return 0;
}
