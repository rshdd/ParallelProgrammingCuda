#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <random>
#include <iostream>
#include <stdio.h>


using namespace std;


__global__ void no_divergence() {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;

    int warp_id = gid / 32;

    if (warp_id % 2 == 0) {
        a = 3.2;
        b = 5.6;
    }
    else {
        a = 3.1416;
        b = 6.666;
    }
}

__global__ void divergence() {
    int gid = blockIdx.x * blockDim.x + threadIdx.x;

    float a, b;

    if (gid % 2 == 0) {
        a = 3.2;
        b = 5.6;
    }
    else {
        a = 3.1416;
        b = 6.666;
    }
}



int main()
{
    int size = 1 << 22;
    dim3 block(128);
    dim3 grid((size * block.x - 1) / block.x);

    no_divergence << <grid, block >> > ();
    hipDeviceSynchronize();

    divergence << <grid, block >> > ();
    hipDeviceSynchronize();

    hipDeviceReset();

    return 0;
}