#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void print_element()
{
    int globalID = (blockIdx.y * blockDim.y + threadIdx.y) * (gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x;
    printf("[DEVICE] ThreadIsx.x: %d\n", globalID);
}

int main()
{
    print_element << <2, 8 >> > ();

    return 0;
}