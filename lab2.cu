#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void addArrays(int* a, int* b, int* c, int* d, int arraySize)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < arraySize) {
        d[idx] = a[idx] + b[idx] + c[idx];
    }
}

int main()
{
    const int arraySize = 10000;

    int *a_cpu, *b_cpu, *c_cpu, *d_cpu;
    int *a_device, *b_device, *c_device, *d_device;

    // Allocate memory for host arrays
    a_cpu = (int*)malloc(arraySize * sizeof(int));
    b_cpu = (int*)malloc(arraySize * sizeof(int));
    c_cpu = (int*)malloc(arraySize * sizeof(int));
    d_cpu = (int*)malloc(arraySize * sizeof(int));

    // Initialize host arrays
    for (int i = 0; i < arraySize; i++) {
        a_cpu[i] = i;
        b_cpu[i] = i;
        c_cpu[i] = i;
    }

    // Allocate memory on the device
    hipMalloc((void**)&a_device, arraySize * sizeof(int));
    hipMalloc((void**)&b_device, arraySize * sizeof(int));
    hipMalloc((void**)&c_device, arraySize * sizeof(int));
    hipMalloc((void**)&d_device, arraySize * sizeof(int));

    // Copy data from host to device
    hipMemcpy(a_device, a_cpu, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_cpu, arraySize * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(c_device, c_cpu, arraySize * sizeof(int), hipMemcpyHostToDevice);

    // Define block and grid dimensions
    int threadsPerBlock = 256;
    int blocksPerGrid = (arraySize + threadsPerBlock - 1) / threadsPerBlock;

    // Launch kernel
    addArrays<<<blocksPerGrid, threadsPerBlock>>>(a_device, b_device, c_device, d_device, arraySize);

    // Copy data from device to host
    hipMemcpy(d_cpu, d_device, arraySize * sizeof(int), hipMemcpyDeviceToHost);

    // Print result
    printf("Vector Resultante:\n");
    for (int i = 0; i < arraySize; ++i) {
        printf("%d\n", d_cpu[i]);
    }

    // Free device memory
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
    hipFree(d_device);

    // Free host memory
    free(a_cpu);
    free(b_cpu);
    free(c_cpu);
    free(d_cpu);

    return 0;
}
