#include "hip/hip_runtime.h"
﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>;

using namespace std;


__global__ void addArrays(int* a, int* b, int* c, int* d)
{
    int hiloX = threadIdx.x;
    int hiloY = threadIdx.y;
    int hiloZ = threadIdx.z;

    int blockX = blockIdx.x;
    int blockY = blockIdx.y;
    int blockZ = blockIdx.z;

    int dimX = blockDim.x;
    int dimY = blockDim.y;
    int dimZ = blockDim.z;

    int globalIDx = blockX * dimX + hiloX;
    int globalIDy = blockY * dimY + hiloY;
    int globalIDz = blockZ * dimZ + hiloZ;

    int gId = (globalIDz * dimX * dimY) + (globalIDy * blockDim.x * gridDim.x) + globalIDx;
    d[gId] = a[gId] + b[gId] + c[gId];
}

int main()
{

    const int arraySize = 10000;

    dim3 blockSize(10, 10, 10);
    dim3 gridSize(10, 10, 2);

    int a_cpu[arraySize];
    int b_cpu[arraySize];
    int c_cpu[arraySize];

    for (int i = 0; i < arraySize; i++) {
        a_cpu[i] = i;
        b_cpu[i] = i;
        c_cpu[i] = i;
    }
    int d_cpu[arraySize];

    int* a_device;
    int* b_device;
    int* c_device;
    int* d_device;

    const int dataCount = arraySize;
    const int data_size = dataCount * sizeof(int);

    // Memory allocation
    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);
    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&d_device, data_size);

    // transfer to GPU memory
    hipMemcpy(a_device, a_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(c_device, c_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(d_device, d_cpu, data_size, hipMemcpyHostToDevice);

    // kernel launch
    addArrays << <gridSize, blockSize >> > (a_device, b_device, c_device, d_device);

    // transfer to CPU host memory from GPU device
    hipMemcpy(c_cpu, c_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_cpu, a_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_cpu, b_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(d_cpu, d_device, data_size, hipMemcpyDeviceToHost);

    printf("\n Vector Resultante: \n");
    for (int i = 0; i < arraySize; ++i) {
        printf("%d \n ", d_cpu[i]);
    }

    hipDeviceReset();
    hipFree(a_device);
    hipFree(b_device);
    hipFree(c_device);
    hipFree(d_device);

    hipDeviceSynchronize();
    return 0;
}
