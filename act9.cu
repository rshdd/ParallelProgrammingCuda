#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

__global__ void stream_test(int* in, int* out, int size) {
    int gid = blockDim.x * blockIdx.x + threadIdx.x;
    if (gid < size)
    {
        // ANY CALC
        for (int i = 0; i < 25; i++) {
            out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
        }
    }
}

int main(int argc, char** argv)
{
    int size = 1 << 18;
    int byte_size = size * sizeof(int);
    int NUM_STREAMS = 10;

    // Host pointers and streams
    int** h_in, ** h_ref;
    hipStream_t* streams;

    // Allocate memory for host pointers and streams
    h_in = (int**)malloc(NUM_STREAMS * sizeof(int*));
    h_ref = (int**)malloc(NUM_STREAMS * sizeof(int*));
    streams = (hipStream_t*)malloc(NUM_STREAMS * sizeof(hipStream_t));

    // Allocate host memory and create streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostMalloc((void**)&h_in[i], byte_size);
        hipHostMalloc((void**)&h_ref[i], byte_size);
        hipStreamCreate(&streams[i]);

        // Initialize host input data
        srand((unsigned int)time(NULL) + i); // Different seed for each stream
        for (int j = 0; j < size; j++) {
            h_in[i][j] = rand();
        }
    }

    // Device pointers
    int** d_in, ** d_out;

    // Allocate memory for device pointers
    d_in = (int**)malloc(NUM_STREAMS * sizeof(int*));
    d_out = (int**)malloc(NUM_STREAMS * sizeof(int*));

    // Allocate device memory
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipMalloc((void**)&d_in[i], byte_size);
        hipMalloc((void**)&d_out[i], byte_size);
    }

    // Kernel Launch and data transfer
    dim3 block(128);
    dim3 grid(size / block.x);

    for (int i = 0; i < NUM_STREAMS; i++) {
        hipMemcpyAsync(d_in[i], h_in[i], byte_size, hipMemcpyHostToDevice, streams[i]);
        stream_test << <grid, block, 0, streams[i] >> > (d_in[i], d_out[i], size);
        hipMemcpyAsync(h_ref[i], d_out[i], byte_size, hipMemcpyDeviceToHost, streams[i]);
    }

    // Synchronize streams
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipStreamSynchronize(streams[i]);
    }

    // Free memory
    for (int i = 0; i < NUM_STREAMS; i++) {
        hipHostFree(h_in[i]);
        hipHostFree(h_ref[i]);
        hipFree(d_in[i]);
        hipFree(d_out[i]);
        hipStreamDestroy(streams[i]);
    }

    // Free allocated arrays
    free(h_in);
    free(h_ref);
    free(d_in);
    free(d_out);
    free(streams);

    return 0;
}