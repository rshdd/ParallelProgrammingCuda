#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>

__global__ void print_global_id()
{
    int globalID = (blockIdx.y * blockDim.y + threadIdx.y) * (gridDim.x * blockDim.x) + blockIdx.x * blockDim.x + threadIdx.x;
    printf("[DEVICE] GlobalId: %d\n", globalID);
}

int main()
{
    dim3 blockSize(4, 2, 1);
    dim3 gridSize(2, 2, 1);
    int* c_cpu;
    int* a_cpu;
    int* b_cpu;

    int* c_device;
    int* a_device;
    int* b_device;
    const int data_count = 10000;
    const int data_size = data_count * sizeof(int);
    c_cpu = (int*)malloc(data_size);
    a_cpu = (int*)malloc(data_size);
    b_cpu = (int*)malloc(data_size);

    // memory allocation
    hipMalloc((void**)&c_device, data_size);
    hipMalloc((void**)&a_device, data_size);
    hipMalloc((void**)&b_device, data_size);

    // transfer CPU host to GPU device
    hipMemcpy(c_device, c_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(a_device, a_cpu, data_size, hipMemcpyHostToDevice);
    hipMemcpy(b_device, b_cpu, data_size, hipMemcpyHostToDevice);

    // launch kernel
    print_global_id << <gridSize, blockSize >> > ();

    // transfer CPU host to GPU device
    hipMemcpy(c_cpu, c_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(a_cpu, a_device, data_size, hipMemcpyDeviceToHost);
    hipMemcpy(b_cpu, b_device, data_size, hipMemcpyDeviceToHost);

    hipDeviceReset();
    hipFree(c_device);
    hipFree(a_device);
    hipFree(b_device);
    return 0;
}
