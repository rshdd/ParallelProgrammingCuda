#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include <time.h>
#include <stdio.h>


__global__ void stream_test(int* in, int* out, int size) {
    int gid = blockDim.x + blockIdx.x + threadIdx.x;
    if (gid < size)
    {
        // ANY CALC
        for (int i = 0; i < 25; i++) {
            out[gid] = in[gid] + (in[gid] - 1) * (gid % 10);
        }
    }
}

int main(int argc, char** argv)
{
    int size = 1 << 18;
    int byte_size = size * sizeof(int);

    // Para poder hacer streams necesitamos hacer pinned memory
    // Initiate host pointer
    int* h_in, * h_ref, * h_in2, * h_ref2;

    hipHostMalloc((void**)&h_in, byte_size);
    hipHostMalloc((void**)&h_ref, byte_size);
    hipHostMalloc((void**)&h_in2, byte_size);
    hipHostMalloc((void**)&h_ref2, byte_size);

    srand((double)time(NULL));
    for (int i = 0; i < size; i++) {
        h_in[i] = rand();
        h_in2[i] = rand();
    }

    // Allocate device pointers
    int* d_in, * d_out, * d_in2, * d_out2;
    hipMalloc((void**)&d_in, byte_size);
    hipMalloc((void**)&d_out, byte_size);
    hipMalloc((void**)&d_in2, byte_size);
    hipMalloc((void**)&d_out2, byte_size);

    // Kernel Launch
    dim3 block(128);
    dim3 grid(size / block.x);
    hipStream_t str, str2;
    hipStreamCreate(&str);
    hipStreamCreate(&str2);

    // Transfer data from host to device (assigning stream)
    hipMemcpyAsync(d_in, h_in, byte_size, hipMemcpyHostToDevice, str);
    // tamaño de memoria compartida, stream (__external__) __shared__
    stream_test << <grid, block, 0, str >> > (d_in, d_out, size);
    hipMemcpyAsync(h_ref, d_out, byte_size, hipMemcpyDeviceToHost, str);

    // Transfer data from host to device (assigning stream)
    hipMemcpyAsync(d_in2, h_in2, byte_size, hipMemcpyHostToDevice, str2);
    // tamaño de memoria compartida, stream (__external__) __shared__
    stream_test << <grid, block, 0, str2 >> > (d_in2, d_out2, size);
    hipMemcpyAsync(h_ref2, d_out2, byte_size, hipMemcpyDeviceToHost, str2);

    return 0;
}