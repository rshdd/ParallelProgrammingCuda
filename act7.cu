#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <time.h>
#include <stdio.h>

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void unrolling2(int* input, int* temp, int size) {
	// En vez de usar 4 bloques, usamos 2.  Reduces bloques a la mitad y aun asi  accedes a los datos (orig size) but with only half 
	int tid = threadIdx.x;
	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 2;
	int index = BLOCK_OFFSET + tid;
	int* i_data = input + BLOCK_OFFSET;

#	// Checar que los threads que se usan son igual o menor al limite de datos Ej si uso Bloque1(32) tid 1 y Bloque2 tid1 es menor que el total de datos, puedo usar ambos bloques, por lo tanto sumo el input
	if ((index + blockDim.x) < size) {
		input[index] += input[index + blockDim.x];
	}

	__syncthreads(); // All should be here, to continue. (order in the court of law)

	// 2 Factor de desdoblamiento To sum all info and put it in index 0
	// CAMBIO PARA EVITAR DIVERGENCIA offset >=32
	for (int offset = blockDim.x / 2; offset >= 32; offset = offset / 2) {
		if (tid < offset) {
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

	// CAMBIO PARA EVITAR DIVERGENCIA
	if (tid < 32) {
		volatile int* vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	// Return sum of everything
	if (tid == 0) {
		temp[blockIdx.x] = i_data[0];
	}
}

__global__ void unrolling4(int* input, int* temp, int size) {
	// We now use 4 blocks (chunks) per 1 block
	int tid = threadIdx.x;
	int BLOCK_OFFSET = blockIdx.x * blockDim.x * 4;
	int index = BLOCK_OFFSET + tid;
	int* i_data = input + BLOCK_OFFSET;

	// Per Grid
	if ((index * 3 * blockDim.x) < size) {
		int a1 = input[index];
		int a2 = input[index + blockDim.x];
		int a3 = input[index + blockDim.x * 2];
		int a4 = input[index + blockDim.x * 3];
	}

	__syncthreads(); // All should be here, to continue. (order in the court of law)

	// Factor de desdoblamiento To sum all info and put it in index 0
	// Per Block we sum
	for (int offset = blockDim.x / 2; offset > 0; offset = offset / 2) {
		if (tid < offset) {
			i_data[tid] += i_data[tid + offset];
		}
		__syncthreads();
	}

	// Return sum of everything
	if (tid == 0) {
		temp[blockIdx.x] = i_data[0];
	}
}

__global__ void unrolling_complete(int* int_array, int* temp_array, int size) {
	int tid = threadIdx.x;

	// element index for this thread
	int index = blockDim.x * blockIdx.x + threadIdx.x;

	// local data pointer
	int* i_data = int_array + blockDim.x * blockIdx.x;

	if (blockDim.x == 1024 && tid < 512)
		i_data[tid] += i_data[tid + 512];
	__syncthreads();

	if (blockDim.x == 512 && tid < 256)
		i_data[tid] += i_data[tid + 256];
	__syncthreads();

	if (blockDim.x == 256 && tid < 128)
		i_data[tid] += i_data[tid + 128];
	__syncthreads();

	if (blockDim.x == 128 && tid < 64)
		i_data[tid] += i_data[tid + 64];
	__syncthreads();

	if (tid < 32) {
		volatile int* vsmem = i_data;
		vsmem[tid] += vsmem[tid + 32];
		vsmem[tid] += vsmem[tid + 16];
		vsmem[tid] += vsmem[tid + 8];
		vsmem[tid] += vsmem[tid + 4];
		vsmem[tid] += vsmem[tid + 2];
		vsmem[tid] += vsmem[tid + 1];
	}

	// Return sum of everything
	if (tid == 0) {
		temp_array[blockIdx.x] = i_data[0];
	}
}

int main(int argc, char** argv) {

	printf("Running parallel reduction with unrolling blocks8 kernel \n");
	int data_size = 1 << 10;
	int byte_size = data_size * sizeof(int);
	int block_size = 32;
	int parallel_reduction = 2;

	int* h_input, * href;
	h_input = (int*)malloc(byte_size);

	for (int i = 0; i < data_size; i++) {
		h_input[i] = (double)(rand() % 10);
	}

	dim3 block(block_size);
	dim3 grid((data_size / block_size) / parallel_reduction);

	printf("Launch parameters -> grid: %d, block: %d \n", grid.x, block.x);

	int temp = sizeof(int) * grid.x;
	h_ref = (int*)malloc(temp);

	int* d_input, * d_temp;
	hipMalloc((void**)&d_input, byte_size);
	hipMalloc((void**)&d_temp, temp);

	hipMemset(d_temp, 0, temp);
	hipMemcpy(d_input, h_input, byte_size, hipMemcpyHostToDevice);

	if (parallel_reduction == 2)
		unrolling2 << < grid, block >> > (d_input, d_temp, data_size);
	else
		unrolling4 << < grid, block >> > (d_input, d_temp, data_size);

	hipDeviceSynchronize();
	hipMemcpy(h_ref, d_temp, temp, hipMemcpyDeviceToHost);

	int gpu_result = 0;
	for (int i = 0; i < grid.x; i++) {
		gpu_result += h_ref[i];
	}

	hipFree(d_input);
	hipFree(d_temp);
	free(h_input);
	free(h_ref);
}